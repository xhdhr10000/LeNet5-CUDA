#include <stdio.h>
#include <stdlib.h>
#include "common.h"
#include "layers/conv.h"
#include "layers/dense.h"
#include "layers/pooling.h"

void test_dense(float *input) {
    Dense d(6, 10);
    d.dump();

    float *doutput = d.forward(input);
    float *output = (float*)malloc(sizeof(float) * 10);
    hipMemcpy(output, doutput, sizeof(float) * 10, hipMemcpyDeviceToHost);

    printf("\nOutput: ");
    for (int i=0; i<10; i++) printf("%9.6lf ", output[i]);
    printf("\n");

    float *loss;
    hipMalloc(&loss, sizeof(float) * 10);
    for (int i=0; i<10; i++) output[i] = 2;
    printf("\nLoss: ");
    for (int i=0; i<10; i++) printf("%9.6lf ", output[i]);
    hipMemcpy(loss, output, sizeof(float) * 10, hipMemcpyHostToDevice);

    d.backward(loss, 0.1);
    d.dump();

    free(output);
    hipFree(loss);
}

void test_pooling(float *input, int c, int h, int w, int s) {
    Pooling p(c, h, w, s);
    p.dump();

    float *doutput = p.forward(input);
    float *output = (float*)malloc(sizeof(float) * c*h/s*w/s);
    hipMemcpy(output, doutput, sizeof(float) * c*h/s*w/s, hipMemcpyDeviceToHost);

    printf("Output:\n");
    for (int i=0; i<c; i++) {
        for (int j=0; j<h/s; j++) {
            for (int k=0; k<w/s; k++) printf("%9.6f ", output[i*h/s*w/s + j*w/s + k]);
            printf("\n");
        }
        printf("\n");
    }

    float *loss;
    hipMalloc(&loss, sizeof(float) * c*h/s*w/s);
    for (int i=0; i<c*h/s*w/s; i++) output[i] = 99;
    printf("\nLoss:\n");
    for (int i=0; i<c; i++) {
        for (int j=0; j<h/s; j++) {
            for (int k=0; k<w/s; k++) printf("%9.6f ", output[i*h*w/s/s + j*w/s + k]);
            printf("\n");
        }
        printf("\n");
    }
    hipMemcpy(loss, output, sizeof(float) * c*h/s*w/s, hipMemcpyHostToDevice);

    float *dd = p.backward(loss, 0.1);
    float *d = (float*)malloc(sizeof(float)*c*h*w);
    hipMemcpy(d, dd, sizeof(float)*c*h*w, hipMemcpyDeviceToHost);

    printf("Output loss:\n");
    for (int i=0; i<c; i++) {
        for (int j=0; j<h; j++) {
            for (int k=0; k<w; k++) printf("%9.6f ", d[i*h*w + j*w + k]);
            printf("\n");
        }
        printf("\n");
    }

    free(d);
    free(output);
    hipFree(loss);
}

void test_conv(float *input, int c, int h, int w, int oc, int k, int s, int p) {
    Conv conv(c, h, w, oc, k, s, p);
    conv.dump();

    int oh = (h+2*p-k)/s+1;
    int ow = (w+2*p-k)/s+1;
    float *doutput = conv.forward(input);
    float *output = (float*)malloc(sizeof(float) * oc*oh*ow);
    hipMemcpy(output, doutput, sizeof(float) * oc*oh*ow, hipMemcpyDeviceToHost);

    printf("Output:\n");
    for (int i=0; i<oc; i++) {
        for (int j=0; j<oh; j++) {
            for (int k=0; k<ow; k++) printf("%9.6f ", output[i*oh*ow + j*ow + k]);
            printf("\n");
        }
        printf("\n");
    }

    float *loss;
    hipMalloc(&loss, sizeof(float) * oc*oh*ow);
    for (int i=0; i<oc*oh*ow; i++) output[i] = 2;
    printf("\nLoss:\n");
    for (int i=0; i<oc; i++) {
        for (int j=0; j<oh; j++) {
            for (int k=0; k<ow; k++) printf("%9.6f ", output[i*oh*ow + j*ow + k]);
            printf("\n");
        }
        printf("\n");
    }
    hipMemcpy(loss, output, sizeof(float) * oc*oh*ow, hipMemcpyHostToDevice);

    float *dd = conv.backward(loss, 0.1);
    conv.dump();
    float *d = (float*)malloc(sizeof(float)*c*h*w);
    hipMemcpy(d, dd, sizeof(float)*c*h*w, hipMemcpyDeviceToHost);

    printf("Output loss:\n");
    for (int i=0; i<c; i++) {
        for (int j=0; j<h; j++) {
            for (int k=0; k<w; k++) printf("%9.6f ", d[i*h*w + j*w + k]);
            printf("\n");
        }
        printf("\n");
    }

    free(d);
    free(output);
    hipFree(loss);
}

int main() {
    rand_init();
    int c = 3, h = 5, w = 5;
    float *input = (float*)malloc(sizeof(float) * c*h*w);
    for (int i=0; i<c*h*w; i++) input[i] = 1;//randn();
    float *dinput;
    hipMalloc(&dinput, sizeof(float) * c*h*w);
    hipMemcpy(dinput, input, sizeof(float) * c*h*w, hipMemcpyHostToDevice);

    printf("Input:\n");
    for (int i=0; i<c; i++) {
        for (int j=0; j<h; j++) {
            for (int k=0; k<w; k++) printf("%9.6lf ", input[i*h*w + j*w + k]);
            printf("\n");
        }
        printf("\n");
    }

    // test_pooling(dinput, c, h, w, 3);
    test_conv(dinput, c, h, w, 5, 3, 2, 2);

    free(input);
    hipFree(dinput);
    return 0;
}
