#include <stdio.h>
#include <stdlib.h>
#include "common.h"
#include "layers/dense.h"
#include "layers/pooling.h"

void test_dense(double *input) {
    Dense d(6, 10);
    d.dump();

    double *doutput = d.forward(input);
    double *output = (double*)malloc(sizeof(double) * 10);
    hipMemcpy(output, doutput, sizeof(double) * 10, hipMemcpyDeviceToHost);

    printf("\nOutput: ");
    for (int i=0; i<10; i++) printf("%9.6lf ", output[i]);
    printf("\n");

    double *loss;
    hipMalloc(&loss, sizeof(double) * 10);
    for (int i=0; i<10; i++) output[i] = 2;
    printf("\nLoss: ");
    for (int i=0; i<10; i++) printf("%9.6lf ", output[i]);
    hipMemcpy(loss, output, sizeof(double) * 10, hipMemcpyHostToDevice);

    d.backward(loss, 0.1);
    d.dump();

    free(output);
    hipFree(loss);
}

void test_pooling(double *input, int c, int h, int w, int s) {
    Pooling p(c, h, w, s);
    p.dump();

    double *doutput = p.forward(input);
    double *output = (double*)malloc(sizeof(double) * c*h/s*w/s);
    hipMemcpy(output, doutput, sizeof(double) * c*h/s*w/s, hipMemcpyDeviceToHost);

    printf("Output:\n");
    for (int i=0; i<c; i++) {
        for (int j=0; j<h/s; j++) {
            for (int k=0; k<w/s; k++) printf("%9.6f ", output[i*h/s*w/s + j*w/s + k]);
            printf("\n");
        }
        printf("\n");
    }

    double *loss;
    hipMalloc(&loss, sizeof(double) * c*h/s*w/s);
    for (int i=0; i<c*h/s*w/s; i++) output[i] = 7;
    printf("\nLoss:\n");
    for (int i=0; i<c; i++) {
        for (int j=0; j<h/s; j++) {
            for (int k=0; k<w/s; k++) printf("%9.6f ", output[i*h*w/s/s + j*w/s + k]);
            printf("\n");
        }
        printf("\n");
    }
    hipMemcpy(loss, output, sizeof(double) * c*h/s*w/s, hipMemcpyHostToDevice);

    double *dd = p.backward(loss, 0.1);
    double *d = (double*)malloc(sizeof(double)*c*h*w);
    hipMemcpy(d, dd, sizeof(double)*c*h*w, hipMemcpyDeviceToHost);

    printf("Output loss:\n");
    for (int i=0; i<c; i++) {
        for (int j=0; j<h; j++) {
            for (int k=0; k<w; k++) printf("%9.6f ", d[i*h*w + j*w + k]);
            printf("\n");
        }
        printf("\n");
    }

    free(d);
    free(output);
    hipFree(dd);
    hipFree(loss);
}

int main() {
    int c = 2, h = 10, w = 10;
    double *input = (double*)malloc(sizeof(double) * c*h*w);
    for (int i=0; i<c*h*w; i++) input[i] = randn();
    double *dinput;
    hipMalloc(&dinput, sizeof(double) * c*h*w);
    hipMemcpy(dinput, input, sizeof(double) * c*h*w, hipMemcpyHostToDevice);

    printf("Input: ");
    for (int i=0; i<c; i++) {
        for (int j=0; j<h; j++) {
            for (int k=0; k<w; k++) printf("%9.6lf ", input[i]);
            printf("\n");
        }
        printf("\n");
    }

    test_pooling(dinput, c, h, w, 2);

    free(input);
    hipFree(dinput);
    return 0;
}
