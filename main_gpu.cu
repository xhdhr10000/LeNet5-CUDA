#include <stdio.h>
#include <stdlib.h>
#include "common.h"
#include "layers/dense.h"

int main() {
    Dense d(6, 10);
    d.dump();

    double *input = (double*)malloc(sizeof(double)*6);
    for (int i=0; i<6; i++) input[i] = 1;
    double *dinput;
    hipMalloc(&dinput, sizeof(double) * 6);
    hipMemcpy(dinput, input, sizeof(double) * 6, hipMemcpyHostToDevice);

    double *doutput = d.forward(dinput);
    double *output = (double*)malloc(sizeof(double) * 10);
    hipMemcpy(output, doutput, sizeof(double) * 10, hipMemcpyDeviceToHost);

    printf("Input: ");
    for (int i=0; i<6; i++) printf("%9.6lf ", input[i]);
    printf("\nOutput: ");
    for (int i=0; i<10; i++) printf("%9.6lf ", output[i]);
    printf("\n");

    double *loss;
    hipMalloc(&loss, sizeof(double) * 10);
    for (int i=0; i<10; i++) output[i] = 2;
    printf("\nLoss: ");
    for (int i=0; i<10; i++) printf("%9.6lf ", output[i]);
    hipMemcpy(loss, output, sizeof(double) * 10, hipMemcpyHostToDevice);

    d.backward(loss, 0.1);
    d.dump();

    free(input);
    free(output);
    hipFree(dinput);
    hipFree(loss);
    return 0;
}
